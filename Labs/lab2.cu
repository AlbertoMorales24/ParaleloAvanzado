#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>;

using namespace std;


__global__ void vectorsSum(int* a, int* b, int* c, int* d)
{

    int globalIDx = blockIdx.x * blockDim.x + threadIdx.x;
    int globalIDy = blockIdx.y * blockDim.y + threadIdx.y;
    int globalIDz = blockIdx.z * blockDim.z + threadIdx.z;

    int globalId = (globalIDz * dimX * dimY) + (globalIDy * blockDim.x * gridDim.x) + globalIDx;

    d[globalId] = a[globalId] + b[globalId] + c[globalId];
}

int main()
{

    const int arraySize = 10000;

    dim3 blockSize(10, 10, 1);
    dim3 gridSize(10, 10, 1);

    int a_host[arraySize];
    int b_host[arraySize];
    int c_host[arraySize];

    int array_final_host[arraySize];

    for (int i = 0; i < arraySize; i++) {
        a_host[i] = i;
        b_host[i] = i;
        c_host[i] = i;
    }

    int* a_device;
    int* b_device;
    int* c_device;

    int* array_final_device;

    const int dataCount = arraySize;
    const int data_size = dataCount * sizeof(int);

    hipMalloc((void**)&a_device, data_size);
    hipMalloc((void**)&b_device, data_size);
    hipMalloc((void**)&c_device, data_size);
    hipMalloc((void**)&array_final_device, data_size);

    hipMemcpy(a_device, a_host, data_size, hipMemcpyHostToDevice);
    hipMemcpy(b_device, b_host, data_size, hipMemcpyHostToDevice);
    hipMemcpy(c_device, c_host, data_size, hipMemcpyHostToDevice);
    hipMemcpy(array_final_device, array_final_host, data_size, hipMemcpyHostToDevice);

    vectorsSum << <gridSize, blockSize >> > (a_device, b_device, c_device, array_final_device);

    hipMemcpy(c_host, c_device, data_size, hipMemcpyDeviceToHost);
    hipMemcpy(a_host, a_device, data_size, hipMemcpyDeviceToHost);
    hipMemcpy(b_host, b_device, data_size, hipMemcpyDeviceToHost);
    hipMemcpy(array_final_host, array_final_device, data_size, hipMemcpyDeviceToHost);

    for (int i = 0; i < arraySize; ++i) {
        printf("%d\n", array_final_host[i]);
    }

    hipDeviceReset();
    hipFree(a_device);
    hipFree(b_device);
    hipFree(c_device);
    hipFree(array_final_device);

    hipDeviceSynchronize();
    
    return 0;
}